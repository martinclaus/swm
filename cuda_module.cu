#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_module.h"

// simple kernel function that adds two vectors
__global__ void vect_add(float *a, float *b, int N)
{
   int idx = threadIdx.x;
   if (idx<N) a[idx] = a[idx] + b[idx];
}


void cu_init_()
{
	if(state == not_initialized) {
		
		// find devices
		int devCount = 0;
		hipGetDeviceCount(&devCount);
        
		if(devCount == 0) {
			state = not_available;
			printf(" Cuda module not available: No device found.\n");
			return;
		}
		
		// use first device
		const int devId = 0;
		hipDeviceProp_t deviceProp;
		
		hipSetDevice(devId);
		hipGetDeviceProperties(&deviceProp, devId);
		
        printf(" Using GPU Device %d: %s with compute capability %d.%d (%d Multiprocessors)\n", 
        		devId, deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
		state = initialized;
	}
}

void cu_testsizes_(size_t *sizeInteger, size_t *sizeReal8)
{
	// check types: fortran <-> C
	if(		*sizeInteger != sizeof(fortInteger)	||
			*sizeReal8   != sizeof(fortReal8)   )
	{
		printf(" Cuda module not available: Size of data types incompatible.\n");
		printf(" \t\tC\tF90\n\tInteger\t%lu\t%lu\n\tReal8\t%lu\t%lu\n", sizeof(fortInteger), *sizeInteger, sizeof(fortReal8), *sizeReal8);
		if(state == initialized)
			cu_finish_();
		
		state = not_available;
	}
}

void cu_setVars_(	fortReal8 *PI, fortReal8 *D2R, fortReal8 *A, fortReal8 *OMEGA,
					fortReal8 *G, fortReal8 *RHO0, fortReal8 *r, fortReal8 *k,
					fortReal8 *Ah, fortReal8 *missval, 
					fortReal8 *gamma_new, fortReal8 *gamma_new_sponge,
					fortReal8 *new_sponge_efolding,
					fortInteger *Nx, fortInteger *Ny, fortInteger *Nt,
					fortReal8 *dt, fortReal8 *dLambda, fortReal8 *dTheta)
{
/*  allocate(u(1:Nx, 1:Ny, 1:Ns))
  allocate(v(1:Nx, 1:Ny, 1:Ns))
  allocate(eta(1:Nx, 1:Ny, 1:Ns))
  allocate(H(1:Nx, 1:Ny))
  allocate(H_u(1:Nx, 1:Ny))
  allocate(H_v(1:Nx, 1:Ny))
  allocate(H_eta(1:Nx, 1:Ny))
  allocate(land_H(1:Nx, 1:Ny))
  allocate(land_u(1:Nx, 1:Ny))
  allocate(land_v(1:Nx, 1:Ny))
  allocate(land_eta(1:Nx, 1:Ny))
  allocate(ocean_H(1:Nx, 1:Ny))
  allocate(ocean_u(1:Nx, 1:Ny))
  allocate(ocean_v(1:Nx, 1:Ny))
  allocate(ocean_eta(1:Nx, 1:Ny))
  allocate(ip1(1:Nx))
  allocate(im1(1:Nx))
  allocate(jp1(1:Ny))
  allocate(jm1(1:Ny))
  allocate(lat_eta(1:Ny))
  allocate(lat_u(1:Ny))
  allocate(lat_v(1:Ny))
  allocate(lat_H(1:Ny))
  allocate(lon_eta(1:Nx))
  allocate(lon_u(1:Nx))
  allocate(lon_v(1:Nx))
  allocate(lon_H(1:Nx))
  allocate(cosTheta_v(1:Ny))
  allocate(cosTheta_u(1:Ny))
  allocate(tanTheta_v(1:Ny))
  allocate(tanTheta_u(1:Ny))		
	 */
}

/*
call CU_setDomain(im1, ip1, jm1, jp1, &
                  lat_u, lat_v, lat_eta, lat_h, &
                  lon_u, lon_v, lon_eta, lon_h, &
                  cosTheta_u, cosTheta_v, &
                  tanTheta_u, tanTheta_v, &
                  H_u, H_v, H_eta, H, &
                  land_u, land_v, land_eta, land_h, &
                  ocean_u, ocean_v, ocean_eta, ocean_h)

*/
void cu_setdomain_()
{
	if(state != initialized)
		return;
	
	// copy
	printf(" Cuda Memcpy took 123ms\n");
}

void cu_advance_()
{
	if(state != initialized)
		return;
	
}

void cu_timestep_()
{
	if(state != initialized)
		return;
	
}

void kernel_wrapper_(float *a, float *b, int *Np)
{
	if(state != initialized)
		return;
	
	float  *a_d, *b_d;  // declare GPU vector copies
   
   int blocks = 1;     // uses 1 block of
   int N = *Np;        // N threads on GPU

   // Allocate memory on GPU
   hipMalloc( (void **)&a_d, sizeof(float) * N );
   hipMalloc( (void **)&b_d, sizeof(float) * N );

   // copy vectors from CPU to GPU
   hipMemcpy( a_d, a, sizeof(float) * N, hipMemcpyHostToDevice );
   hipMemcpy( b_d, b, sizeof(float) * N, hipMemcpyHostToDevice );

   // call function on GPU
   vect_add<<< blocks, N >>>( a_d, b_d, N);

   // copy vectors back from GPU to CPU
   hipMemcpy( a, a_d, sizeof(float) * N, hipMemcpyDeviceToHost );
   hipMemcpy( b, b_d, sizeof(float) * N, hipMemcpyDeviceToHost );

   // free GPU memory
   hipFree(a_d);
   hipFree(b_d);
   return;
}


void cu_copytohost_u_(fortReal8 *h_u)
{
	printf( " called copyToHost_u() with %d = %f\n", h_u, h_u[0]);
}

void cu_copytohost_v_(fortReal8 *h_v)
{
	printf( " called copyToHost_v()with %d = %f\n", h_v, h_v[0]);
}

void cu_copytohost_eta_(fortReal8 *h_eta)
{
	printf( " called copyToHost_eta() with %d = %f\n", h_eta, h_eta[0]);
}

void cu_finish_()
{
	if(state == initialized) {
		printf(" Freeing GPU resources\n");
		// de allocate
		state = not_initialized;
	}
}
