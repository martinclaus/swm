#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_module.h"
#include "cuda_memory.h"
#include "cuda_kernels.h"
#include "model.h"

//=============================================================================
enum module_state {
		not_initialized, 	// didn't try
		initialized,		// tried and succeeded
		not_available		// tried and failed
} state;

//=============================================================================
dim3 grid, blocksize;
hipStream_t stream[3];

//=============================================================================
// Testing device capabilities
void cu_init_()
{
	// find devices
	int devCount = 0;
	hipGetDeviceCount(&devCount);

	if(devCount == 0) {
		state = not_available;
		printf(" Cuda module not available: No device found.\n");
		return;
	}

	// use first device
	const int devId = 0;
	hipDeviceProp_t deviceProp;

	hipSetDevice(devId);
	hipGetDeviceProperties(&deviceProp, devId);

	if(deviceProp.major <= 1 || (deviceProp.major <= 1 && deviceProp.minor < 3)) {
		state = not_available;
		printf(" Cuda Error: compute capability of GPU < 1.3 : No support for double precision.\n");
		return;
	}
	
	printf(" Using GPU Device %d: %s with compute capability %d.%d (%d Multiprocessors)\n", 
		devId, deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

	// Create execution streams
#ifdef CUDA_USE_STREAMS
	for(int i = 0; i < 3; i++) {
		hipStreamCreate(&stream[i]);
	}
#endif
		
	state = initialized;	
}

//=============================================================================
// The grid is fitted to the blocksize. If the grid size is not a multiple of the blocksize, it is padded.
// This allows for use of the optimal block size and also odd grid dimensions. The extra elements of the grid
// are not computed by the kernel.
void fitGridSize() {
	// Set kernel size
	blocksize = dim3(8, 4);
	
	// Padding
	int kernelNx = hConstParams.Nx;
	if(hConstParams.Nx % blocksize.x != 0)
	  kernelNx = (hConstParams.Nx / blocksize.x + 1) * blocksize.x;
	
	int kernelNy = hConstParams.Ny;
	if(kernelNy % blocksize.y != 0)
	  kernelNy = (hConstParams.Ny / blocksize.y + 1) * blocksize.y;
	
	grid = dim3(kernelNx / blocksize.x, kernelNy / blocksize.y);
	
	// be verbose
	if(kernelNx == hConstParams.Nx && kernelNy == hConstParams.Ny)
		printf(" Cuda_module: No Padding necessary.\n");
	else
		printf(" Cuda_module: Padded to size %dx%d.\n", kernelNx, kernelNy);
}

//=============================================================================
void cu_setconstants_(fortInteger *pNx, fortInteger *pNy, fortReal8 *pDt, fortReal8 *pAB_C1, fortReal8 *pAB_C2, fortReal8 *A, fortReal8 *dTheta, fortReal8 *dLambda)
{	
	if(state != initialized)
		return;
	
	// Set initial values
	hVarParams.N0 = 0;
	hVarParams.N0p1 = 1;
	hVarParams.NG0 = 1;
	hVarParams.NG0m1 = 0;

	// Set constants
	hConstParams.Nx = *pNx;
	hConstParams.Ny = *pNy;
	hConstParams.dt = *pDt;
	hConstParams.AB_C1 = *pAB_C1;
	hConstParams.AB_C2 = *pAB_C2;
	hConstParams.A = *A;
	hConstParams.dTheta = *dTheta;
	hConstParams.dLambda = *dLambda;

	fitGridSize();
}

//=============================================================================
void cu_setfields_(
		bool *ocean_eta, bool *ocean_u, bool *ocean_v,
		fortReal8 *SWM_Coef_eta, fortReal8 *SWM_Coef_u, fortReal8 *SWM_Coef_v,
		fortReal8 *impl_eta, fortReal8 *impl_u, fortReal8 *impl_v,
		fortReal8 *G_eta, fortReal8 *G_u, fortReal8 *G_v,
		fortReal8 *SWM_eta, fortReal8 *SWM_u, fortReal8 *SWM_v,
		fortReal8 *F_eta, fortReal8 *F_x, fortReal8 *F_y,
		fortReal8 *H_u, fortReal8 *H_v,
		fortReal8 *cos_lat_v
		)
{	
	if(state != initialized)
		return;
	
	if(hConstParams.Nx <= 0 || hConstParams.Ny <= 0) {
	  printf(" Cuda module: Warning: Nx (%d) or Ny (%d) invalid.\n", hConstParams.Nx, hConstParams.Ny);
	  return;
	}
	
 	cuAllocateDataMemory();
	
	const size_t elemcount = hConstParams.Nx * hConstParams.Ny;
	const size_t sizeInts = elemcount * sizeof(fortInteger);
	const size_t sizeReals = elemcount * sizeof(fortReal8);
		
	// Convert ocean field from boolean to integers
	fortInteger *tmp_ocean = (fortInteger *) malloc(sizeInts);
	
	for(int i = 0; i < elemcount; i++)
	  tmp_ocean[i] = ocean_eta[i];
	checkCudaErrors( hipMemcpy(hDevMem.ocean_eta, tmp_ocean, sizeInts, hipMemcpyHostToDevice) );
	
	for(int i = 0; i < elemcount; i++)
	  tmp_ocean[i] = ocean_u[i];
	checkCudaErrors( hipMemcpy(hDevMem.ocean_u,   tmp_ocean, sizeInts, hipMemcpyHostToDevice) );
	
	for(int i = 0; i < elemcount; i++)
	  tmp_ocean[i] = ocean_v[i];
	checkCudaErrors( hipMemcpy(hDevMem.ocean_v,   tmp_ocean, sizeInts, hipMemcpyHostToDevice) );
	
	free(tmp_ocean);
		
	// Copy the data to the device
	checkCudaErrors( hipMemcpy(hDevMem.SWM_Coef_eta, SWM_Coef_eta, sizeReals * 9,  hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.SWM_Coef_u,   SWM_Coef_u,   sizeReals * 11, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.SWM_Coef_v,   SWM_Coef_v,   sizeReals * 11, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipMemcpy(hDevMem.impl_eta, impl_eta, sizeReals, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.impl_u,   impl_u,   sizeReals, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.impl_v,   impl_v,   sizeReals, hipMemcpyHostToDevice) );

	checkCudaErrors( hipMemcpy(hDevMem.G_eta, G_eta, sizeReals * 2, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.G_u,   G_u,   sizeReals * 2, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.G_v,   G_v,   sizeReals * 2, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipMemcpy(hDevMem.SWM_eta, SWM_eta, sizeReals * 2, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.SWM_u,   SWM_u,   sizeReals * 2, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.SWM_v,   SWM_v,   sizeReals * 2, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipMemcpy(hDevMem.F_eta, F_eta, sizeReals, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.F_x,   F_x,   sizeReals, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.F_y,   F_y,   sizeReals, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipMemcpy(hDevMem.H_u,   H_u,   sizeReals, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.H_v,   H_v,   sizeReals, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipMemcpy(hDevMem.cos_lat_v,   cos_lat_v,   hConstParams.Ny * sizeof(fortReal8), hipMemcpyHostToDevice) );
}

//=============================================================================
void cu_setforcing_(fortReal8* F_x, fortReal8* F_y, fortReal8* F_eta)
{
	if(state != initialized)
		return;

	if(hDevMem.F_eta == 0 || hDevMem.F_x == 0 || hDevMem.F_y == 0)
		printf(" Cuda module: Warning: forcing fields not allocated in GPU memory.\n");
	
	const size_t fieldsize = hConstParams.Nx * hConstParams.Ny * sizeof(fortReal8);
#ifdef CUDA_USE_STREAMS
	checkCudaErrors( hipMemcpyAsync(hDevMem.F_eta, F_eta, fieldsize, hipMemcpyHostToDevice, stream[0]) );
	checkCudaErrors( hipMemcpyAsync(hDevMem.F_x,   F_x,   fieldsize, hipMemcpyHostToDevice, stream[1]) );
	checkCudaErrors( hipMemcpyAsync(hDevMem.F_y,   F_y,   fieldsize, hipMemcpyHostToDevice, stream[2]) );
#else
	checkCudaErrors( hipMemcpy(hDevMem.F_eta, F_eta, fieldsize, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.F_x,   F_x,   fieldsize, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(hDevMem.F_y,   F_y,   fieldsize, hipMemcpyHostToDevice) );	
#endif
}

//=============================================================================
void cu_advance_()
{	
	// flipflop
	hVarParams.N0 = (hVarParams.N0 + 1) % 2;
	hVarParams.N0p1 = (hVarParams.N0p1 + 1) % 2;
	hVarParams.NG0 = (hVarParams.NG0 + 1) % 2;
	hVarParams.NG0m1 = (hVarParams.NG0m1 + 1) % 2;
}

//=============================================================================
void cu_timestep_()
{
	if(state != initialized)
		return;
	
	static int itt = 0;
		
	if(state != initialized)
		return;
	
	if(itt++ < 2) {
		hipDeviceSynchronize();
		cuExecuteEulerForwardKernel(grid, blocksize, stream);
	} else {
		hipDeviceSynchronize();
		cuExecuteAdamBashForthKernel(grid, blocksize, stream);
	}
}

//=============================================================================
void cu_computestreamfunction_()
{
	if(state != initialized)
		return;

	hipDeviceSynchronize(); // wait until calculation of u and v have finished
#ifdef  CALC_LIB_ELLIPTIC_SOLVER
	printf("Cuda Warning: CALC_LIB_ELLIPCTIC_SOLVER is defined but is is not implemented yet.\n");
#endif
	cuExecuteStreamFunctionKernel(grid, blocksize, stream);
}

//=============================================================================
void cu_copytohost_eta_(fortReal8 *h_eta)
{
	if(state != initialized)
		return;
	
	size_t sizeReals = hConstParams.Nx*hConstParams.Ny*sizeof(fortReal8);
#ifdef CUDA_USE_STREAMS
 	hipStreamSynchronize(stream[0]); // wait for stream to complete calculation of eta
	checkCudaErrors( hipMemcpyAsync(h_eta, hDevMem.SWM_eta, sizeReals * 2, hipMemcpyDeviceToHost, stream[0]) );
#else
	checkCudaErrors( hipMemcpy(h_eta, hDevMem.SWM_eta, sizeReals * 2, hipMemcpyDeviceToHost) );
#endif
}

//=============================================================================
void cu_copytohost_u_(fortReal8 *h_u)
{
	if(state != initialized)
		return;
	
	size_t sizeReals = hConstParams.Nx*hConstParams.Ny*sizeof(fortReal8);
#ifdef CUDA_USE_STREAMS
	hipStreamSynchronize(stream[1]); // wait for stream to complete calculation of u
	checkCudaErrors( hipMemcpyAsync(h_u,   hDevMem.SWM_u,   sizeReals * 2, hipMemcpyDeviceToHost, stream[1]) );
#else
	checkCudaErrors( hipMemcpy(h_u, hDevMem.SWM_u, sizeReals * 2, hipMemcpyDeviceToHost) );
#endif
}

//=============================================================================
void cu_copytohost_v_(fortReal8 *h_v)
{
	if(state != initialized)
		return;
	
	size_t sizeReals = hConstParams.Nx*hConstParams.Ny*sizeof(fortReal8);
#ifdef CUDA_USE_STREAMS
	hipStreamSynchronize(stream[2]); // wait for stream to complete calculation of v
	checkCudaErrors( hipMemcpyAsync(h_v,   hDevMem.SWM_v,   sizeReals * 2, hipMemcpyDeviceToHost, stream[2]) );
#else
	checkCudaErrors( hipMemcpy(h_v, hDevMem.SWM_v, sizeReals * 2, hipMemcpyDeviceToHost) );
#endif
}

//=============================================================================
void cu_copytohost_psi_(fortReal8 *h_psi)
{
	if(state != initialized)
		return;
	
	size_t sizeReals = hConstParams.Nx*hConstParams.Ny*sizeof(fortReal8);
	hipDeviceSynchronize(); // wait for all kernels to complete execution
	checkCudaErrors( hipMemcpy(h_psi, hDevMem.diag_psi, sizeReals, hipMemcpyDeviceToHost) );
}

//=============================================================================
void cu_finish_()
{	
	if(state == initialized) 
	{
		printf(" Freeing GPU resources\n");
#ifdef CUDA_USE_STREAMS
		for(int i = 0; i < 3; i++) {
			hipStreamDestroy(stream[i]);
		}
#endif		
		cuFreeDataMemory();		
		// de allocate
		state = not_initialized;
	}
}

//=============================================================================
void cu_testsizes_(size_t *sizeInteger, size_t *sizeReal8)
{
	printf(" Cuda_module: cu_testsizes_()\n");
	// check types: fortran <-> C
	if(	*sizeInteger != sizeof(fortInteger)	 ||
		*sizeReal8   != sizeof(fortReal8)   )
	{
		printf(" Cuda module not available: Size of data types incompatible.\n");
		printf(" \t\tC\tF90\n\tInteger\t%lu\t%lu\n\tReal8\t%lu\t%lu\n", sizeof(fortInteger), *sizeInteger, sizeof(fortReal8), *sizeReal8);
		
		if(state == initialized)
			cu_finish_();
		
		state = not_available;
	}
	
#ifdef QUADRATIC_BOTTOM_FRICTION
	printf(" Warning: Cuda module uses linear bottom friction but quadratic friction is defined");
#endif	
}