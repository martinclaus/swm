#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include "cuda_memory.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "model.h"

//================== Kernel Declaration: AdamBashforth, Euler, Psi ==============
__global__ void adamBashforth_eta(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void adamBashforth_u(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void adamBashforth_v(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);

__global__ void eulerForward_eta(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void eulerForward_u(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void eulerForward_v(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);

__global__ void computeStreamFunctionA(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void computeStreamFunctionB(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);
__global__ void computeStreamFunctionC(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem);


//=============================================================================
void cuExecuteAdamBashForthKernel(dim3 const grid, dim3 const blocksize, hipStream_t const *stream) {
#ifdef CUDA_USE_STREAMS
	adamBashforth_eta<<< grid, blocksize, 0, stream[0] >>>(hConstParams, hVarParams, hDevMem);
	adamBashforth_u  <<< grid, blocksize, 0, stream[1] >>>(hConstParams, hVarParams, hDevMem);
	adamBashforth_v  <<< grid, blocksize, 0, stream[2] >>>(hConstParams, hVarParams, hDevMem);
#else
	adamBashforth_eta<<< grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	adamBashforth_u  <<< grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	adamBashforth_v  <<< grid, blocksize>>>(hConstParams, hVarParams, hDevMem);	
#endif
}

//=============================================================================
void cuExecuteEulerForwardKernel(dim3 const grid, dim3 const blocksize, hipStream_t const *stream) {
#ifdef CUDA_USE_STREAMS
	eulerForward_eta<<<grid, blocksize, 0, stream[0] >>>(hConstParams, hVarParams, hDevMem);
	eulerForward_u<<<grid, blocksize, 0, stream[1] >>>(hConstParams, hVarParams, hDevMem);
	eulerForward_v<<<grid, blocksize, 0, stream[2] >>>(hConstParams, hVarParams, hDevMem);
#else
	eulerForward_eta<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	eulerForward_u<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	eulerForward_v<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
#endif	
}

//=============================================================================
void cuExecuteStreamFunctionKernel(dim3 const grid, dim3 const blocksize, hipStream_t const *streams)
{
	hipDeviceSynchronize();
	computeStreamFunctionA<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	computeStreamFunctionB<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
	computeStreamFunctionC<<<grid, blocksize>>>(hConstParams, hVarParams, hDevMem);
}

//=============================================================================
// Makros to access arrays in fortran style
// Array access is 'circular', meaning index violations non-existent
// Constant input:
#define ocean_eta(i,j) 		devMem.ocean_eta[((j) % Ny) * Nx + ((i) % Nx)]
#define ocean_u(i,j) 		devMem.ocean_u[((j) % Ny) * Nx + ((i) % Nx)]
#define ocean_v(i,j) 		devMem.ocean_v[((j) % Ny) * Nx + ((i) % Nx)]
#define impl_eta(i,j)		devMem.impl_eta[((j) % Ny) * Nx + ((i) % Nx)]
#define impl_u(i,j)		devMem.impl_u[((j) % Ny) * Nx + ((i) % Nx)]
#define impl_v(i,j)		devMem.impl_v[((j) % Ny) * Nx + ((i) % Nx)]
#define SWM_Coef_eta(i,j,k) 	devMem.SWM_Coef_eta[(k) * Nx * 9 + ((j) % Ny) * 9 + (i)]
#define SWM_Coef_u(i,j,k) 	devMem.SWM_Coef_u[(k) * Nx * 11 + ((j) % Ny) * 11 + (i)]
#define SWM_Coef_v(i,j,k) 	devMem.SWM_Coef_v[(k) * Nx * 11 + ((j) % Ny) * 11 + (i)]
#define cos_lat_v(j)		devMem.cos_lat_v[j % Ny]
#define H_v(i,j)		devMem.H_v[((j) % Ny) * Nx + ((i) % Nx)]
#define H_u(i,j)		devMem.H_u[((j) % Ny) * Nx + ((i) % Nx)]

// Externally changed input:
#define F_eta(i,j) 		devMem.F_eta[((j) % Ny) * Nx + ((i) % Nx)]
#define F_x(i,j) 		devMem.F_x[((j) % Ny) * Nx + ((i) % Nx)]
#define F_y(i,j) 		devMem.F_y[((j) % Ny) * Nx + ((i) % Nx)]

// Cuda Output:
#define G_eta(i,j,k) 		devMem.G_eta[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define G_u(i,j,k) 		devMem.G_u[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define G_v(i,j,k) 		devMem.G_v[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define SWM_eta(i,j,k)		devMem.SWM_eta[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define SWM_u(i,j,k) 		devMem.SWM_u[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define SWM_v(i,j,k) 		devMem.SWM_v[(k) * Nx * Ny + ((j) % Ny) * Nx + ((i) % Nx)]
#define psi(i,j)		devMem.diag_psi[((j) % Ny) * Nx + ((i) % Nx)]

//=============================================================================
__global__ void adamBashforth_eta(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
	const unsigned int NG0m1 = varParams.NG0m1;
	
	if( i < Nx && j < Ny && ocean_eta(i, j) == 1 )
	{
		G_eta(i, j, NG0) = 
				SWM_eta(i, j, N0) 	* SWM_Coef_eta(0,i,j) +
				SWM_eta(i + 1, 	j, N0)	* SWM_Coef_eta(1,i,j) +
				SWM_eta(i - 1, 	j, N0) 	* SWM_Coef_eta(2,i,j) +
				SWM_eta(i, j + 1, N0) 	* SWM_Coef_eta(3,i,j) +
				SWM_eta(i, j - 1, N0) 	* SWM_Coef_eta(4,i,j) +
				SWM_u(	i + 1, 	j, N0) 	* SWM_Coef_eta(5,i,j) +
				SWM_u(	i, j, N0) 	* SWM_Coef_eta(6,i,j) +
				SWM_v(	i, j + 1, N0) 	* SWM_Coef_eta(7,i,j) +
				SWM_v(	i, j, N0) 	* SWM_Coef_eta(8,i,j) +
				F_eta(	i, j);
		  
		SWM_eta(i, j, N0p1) =
				(SWM_eta(i, j, N0) + constParams.dt * (constParams.AB_C1 * G_eta(i, j, NG0) - constParams.AB_C2 * G_eta(i, j, NG0m1)))/impl_eta(i,j);
	}
}

//=============================================================================
__global__ void adamBashforth_u(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
	const unsigned int NG0m1 = varParams.NG0m1;
    
	if( i < Nx && j < Ny && ocean_u(i, j) == 1 )
	{
		G_u(i, j, NG0) = 
				SWM_u(	i, 	j, N0) 		* SWM_Coef_u(0, i, j) +
				SWM_u(	i + 1, 	j, N0) 		* SWM_Coef_u(1, i, j) +
				SWM_u(	i - 1, 	j, N0) 		* SWM_Coef_u(2, i, j) +
				SWM_u(	i, 	j + 1, N0) 	* SWM_Coef_u(3, i, j) +
				SWM_u(	i, 	j - 1, N0) 	* SWM_Coef_u(4, i, j) +
				SWM_v(	i, 	j, N0) 		* SWM_Coef_u(5, i, j) +
				SWM_v(	i - 1,	j, N0) 		* SWM_Coef_u(6, i, j) +
				SWM_v(	i - 1, 	j + 1, N0) 	* SWM_Coef_u(7, i, j) +
				SWM_v(	i, 	j + 1, N0) 	* SWM_Coef_u(8, i, j) +
				SWM_eta(i, 	j, N0)		* SWM_Coef_u(9,i, j) +
				SWM_eta(i - 1, 	j, N0)		* SWM_Coef_u(10,i, j) +
				F_x(i, j);
		
		SWM_u(i, j, N0p1) =
				(SWM_u(i, j, N0) + constParams.dt * (constParams.AB_C1 * G_u(i, j, NG0) - constParams.AB_C2 * G_u(i, j, NG0m1)))/impl_u(i,j);
	}
}

//=============================================================================
__global__ void adamBashforth_v(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
	const unsigned int NG0m1 = varParams.NG0m1;
  
	if( i < Nx && j < Ny && ocean_v(i, j) == 1 )
	{
		G_v(i, j, NG0) = 
				SWM_v(	i, 	j, N0) 		* SWM_Coef_v(0, i, j) +
				SWM_v(	i + 1, 	j, N0) 		* SWM_Coef_v(1, i, j) +
				SWM_v(	i - 1, 	j, N0) 		* SWM_Coef_v(2, i, j) +
				SWM_v(	i, 	j + 1, N0) 	* SWM_Coef_v(3, i, j) +
				SWM_v(	i, 	j - 1, N0) 	* SWM_Coef_v(4, i, j) +
				SWM_u(	i + 1, 	j - 1, N0) 	* SWM_Coef_v(5, i, j) +
				SWM_u(	i, 	j - 1, N0) 	* SWM_Coef_v(6, i, j) +
				SWM_u(	i, 	j, N0) 		* SWM_Coef_v(7, i, j) +
				SWM_u(	i + 1, 	j, N0) 		* SWM_Coef_v(8, i, j) +
				SWM_eta(i, 	j, N0)		* SWM_Coef_v(9,i, j) +
				SWM_eta(i, 	j - 1, N0)	* SWM_Coef_v(10,i, j) +
				F_y(i, j);
		
		SWM_v(i, j, N0p1) =
				(SWM_v(i, j, N0) + constParams.dt * (constParams.AB_C1 * G_v(i, j, NG0) - constParams.AB_C2 * G_v(i, j, NG0m1)))/impl_v(i,j);
	}
}

//=============================================================================
// Kernels for euler forward iteration
//=============================================================================
__global__ void eulerForward_eta(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
    
	if( i < Nx && j < Ny && ocean_eta(i, j) == 1 )
	{
		G_eta(i, j, NG0) = 
				SWM_eta(i, j, N0) 	* SWM_Coef_eta(0,i,j) +
				SWM_eta(i + 1, j, N0) 	* SWM_Coef_eta(1,i,j) +
				SWM_eta(i - 1, j, N0) 	* SWM_Coef_eta(2,i,j) +
				SWM_eta(i, j + 1, N0)	* SWM_Coef_eta(3,i,j) +
				SWM_eta(i, j - 1, N0) 	* SWM_Coef_eta(4,i,j) +
				SWM_u(i + 1, j, N0) 	* SWM_Coef_eta(5,i,j) +
				SWM_u(i, j, N0)		* SWM_Coef_eta(6,i,j) +
				SWM_v(i, j + 1, N0) 	* SWM_Coef_eta(7,i,j) +
				SWM_v(i, j, N0) 	* SWM_Coef_eta(8,i,j) +
				F_eta(i, j);
		
		SWM_eta(i, j, N0p1) = (SWM_eta(i, j, N0) + constParams.dt * G_eta(i, j, NG0))/impl_eta(i,j);
	}
}

//=============================================================================
__global__ void eulerForward_u(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
    
	if( i < Nx && j < Ny && ocean_u(i, j) == 1 )
	{
		G_u(i, j, NG0) = 
				SWM_u(i, j, N0) 	* SWM_Coef_u(0,i,j) +
				SWM_u(i + 1, j, N0) 	* SWM_Coef_u(1,i,j) +
				SWM_u(i - 1, j, N0) 	* SWM_Coef_u(2,i,j) +
				SWM_u(i, j + 1, N0)	* SWM_Coef_u(3,i,j) +
				SWM_u(i, j - 1, N0) 	* SWM_Coef_u(4,i,j) +
				SWM_v(i, j, N0) 	* SWM_Coef_u(5,i,j) +
				SWM_v(i - 1,j, N0) 	* SWM_Coef_u(6,i,j) +
				SWM_v(i - 1, j + 1, N0) * SWM_Coef_u(7,i,j) +
				SWM_v(i, j + 1, N0) 	* SWM_Coef_u(8,i,j) +
				SWM_eta(i, j, N0) 	* SWM_Coef_u(9,i,j) +
				SWM_eta(i - 1, j, N0) 	* SWM_Coef_u(10,i,j) +
				F_x(i, j);
		
		SWM_u(i, j, N0p1) = (SWM_u(i, j, N0) + constParams.dt * G_u(i, j, NG0))/impl_u(i,j);
	}
}    

//=============================================================================
__global__ void eulerForward_v(const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	const unsigned int N0 = varParams.N0;
	const unsigned int N0p1 = varParams.N0p1;
	const unsigned int NG0 = varParams.NG0;
    
	if( i < Nx && j < Ny && ocean_v(i, j) == 1 )
	{
		G_v(i, j, NG0) = 
				SWM_v(i, j, N0) 	* SWM_Coef_v(0,i,j) +
				SWM_v(i + 1, j, N0) 	* SWM_Coef_v(1,i,j) +
				SWM_v(i - 1, j, N0) 	* SWM_Coef_v(2,i,j) +
				SWM_v(i, j + 1, N0)	* SWM_Coef_v(3,i,j) +
				SWM_v(i, j - 1, N0) 	* SWM_Coef_v(4,i,j) +
				SWM_u(i + 1, j - 1, N0) * SWM_Coef_v(5,i,j) +
				SWM_u(i,j - 1, N0) 	* SWM_Coef_v(6,i,j) +
				SWM_u(i, j, N0) 	* SWM_Coef_v(7,i,j) +
				SWM_u(i + 1, j, N0) 	* SWM_Coef_v(8,i,j) +
				SWM_eta(i, j, N0) 	* SWM_Coef_v(9,i,j) +
				SWM_eta(i, j - 1, N0) 	* SWM_Coef_v(10,i,j) +
				F_y(i, j);
		
		SWM_v(i, j, N0p1) = (SWM_v(i, j, N0) + constParams.dt * G_v(i, j, NG0))/impl_v(i,j);
	}
}
//=============================================================================

//=============================================================================
__global__ void computeStreamFunctionA(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	
	if( j == 0 && i < Nx - 1)
	{
		psi(i,j) = 0;
		for(int x = i; x < Nx - 1; x++) {
			psi(i,j) += 
#ifdef BAROTROPIC
					H_v(x, j) *
#endif
					ocean_v(x, j) * SWM_v(x, j, varParams.N0p1);    
		}
		psi(i,j) *= (-1) * constParams.A * cos_lat_v(j) * constParams.dLambda;
	}
}

//=============================================================================
__global__ void computeStreamFunctionB(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	
	if( i == Nx-1 && j < Ny && j > 0 )
	{
		psi(i,j) = 0;
		for(int y = 0; y < j; y++) {
			psi(i,j) += 
#ifdef BAROTROPIC
					H_u(i, y) *
#endif
					ocean_u(i, y) * SWM_u(i, y, varParams.N0p1);    
		}
		psi(i,j) *= (-1) * constParams.A * constParams.dTheta;
	}
}

//=============================================================================
__global__ void computeStreamFunctionC(  const constExecutionParameters constParams, const varExecutionParameters varParams, const devMemory devMem)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int Nx = constParams.Nx;
	const unsigned int Ny = constParams.Ny;
	
	if( i < Nx-1 && j < Ny && j > 0 )
	{
		double sum1 = 0;
		for(int x = i; x < Nx-1; x++) {
			sum1 +=
#ifdef BAROTROPIC
					H_v(x, j) *
#endif
					SWM_v(x, j, varParams.N0p1);
		}
		sum1 *= (-1) * constParams.A * cos_lat_v(j) * constParams.dLambda;
				
		double sum2 = 0;
		for(int y = 0; y < j-1; y++) 	{
			sum2 +=
#ifdef BAROTROPIC
					H_u(i, y) *
#endif
					SWM_u(i, y, varParams.N0p1);
		}
		sum2 *= constParams.A * constParams.dTheta;
		
		psi(i,j) = (sum1 + psi(Nx-1, j) - sum2 + psi(i,0)) / 2.;
	}
}