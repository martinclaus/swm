#include "cuda_memory.h"
#include "stdio.h"
#include "cuda_kernels.h"
#include "model.h"

constExecutionParameters hConstParams;
varExecutionParameters hVarParams;
devMemory hDevMem;

//=============================================================================
void cuAllocateDataMemory()
{
	printf(" Cuda_module: cu_init_mem_()\n");
	
	if(hConstParams.Nx <= 0 || hConstParams.Ny <= 0)
		printf(" Cuda module: Warning: Nx (%d) or Ny (%d) invalid.\n", hConstParams.Nx, hConstParams.Ny);
	
	const size_t elemcount = hConstParams.Nx * hConstParams.Ny;
	const size_t sizeInts = elemcount * sizeof(fortInteger);
	const size_t sizeReals = elemcount * sizeof(fortReal8);
	
	// Allocate device memory
	checkCudaErrors( hipMalloc((void **)&hDevMem.ocean_eta, 	sizeInts) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.ocean_u,   	sizeInts) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.ocean_v,   	sizeInts) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_Coef_eta, 	sizeReals * 9) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_Coef_u,   	sizeReals * 11) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_Coef_v,   	sizeReals * 11) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.impl_eta, 	sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.impl_u,   	sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.impl_v,   	sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.G_eta, 		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.G_u,   		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.G_v,   		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_eta, 		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_u,   		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.SWM_v,   		sizeReals * 2) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.diag_psi,		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.F_eta, 		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.F_x,   		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.F_y,   		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.H_u,   		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.H_v,   		sizeReals) );
	checkCudaErrors( hipMalloc((void **)&hDevMem.cos_lat_v, 	sizeof(fortReal8) * hConstParams.Ny) );
}

void cuFreeDataMemory()
{
	hipDeviceSynchronize(); // wait for other tasks to finish
	hipFree(hDevMem.ocean_eta);
	hipFree(hDevMem.ocean_u);
	hipFree(hDevMem.ocean_v);
	hipFree(hDevMem.SWM_Coef_eta);
	hipFree(hDevMem.SWM_Coef_u);
	hipFree(hDevMem.SWM_Coef_v);
	hipFree(hDevMem.impl_eta);
	hipFree(hDevMem.impl_u);
	hipFree(hDevMem.impl_v);
	hipFree(hDevMem.G_eta);
	hipFree(hDevMem.G_u);
	hipFree(hDevMem.G_v);
	hipFree(hDevMem.SWM_eta);
	hipFree(hDevMem.SWM_u);
	hipFree(hDevMem.SWM_v);
	hipFree(hDevMem.diag_psi);
	hipFree(hDevMem.F_eta);
	hipFree(hDevMem.F_x);
	hipFree(hDevMem.F_y);
	hipFree(hDevMem.H_u);
	hipFree(hDevMem.H_v);
	hipFree(hDevMem.cos_lat_v);
}


void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
	if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}
