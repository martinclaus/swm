#include "hip/hip_runtime.h"
//=============================================================================
// Device memory: Put execution parameters into special constant memory region for faster access
// Const Parameters
__constant__  fortReal8 dc_dt;
__constant__  fortReal8 dc_AB_C1;
__constant__  fortReal8 dc_AB_C2;
__constant__  fortInteger dc_Nx;
__constant__  fortInteger dc_Ny;

// Variable Parameters
__constant__  int dc_NG0;
__constant__  int dc_NG0m1;
__constant__  int dc_N0;
__constant__  int dc_N0p1;

// DevMem
__constant__  fortInteger *dc_ocean_eta;
__constant__  fortInteger *dc_ocean_u;
__constant__  fortInteger *dc_ocean_v;
__constant__  fortReal8 *dc_SWM_Coef_eta;
__constant__  fortReal8 *dc_SWM_Coef_u;
__constant__  fortReal8 *dc_SWM_Coef_v;
__constant__  fortReal8 *dc_impl_eta;
__constant__  fortReal8 *dc_impl_u;
__constant__  fortReal8 *dc_impl_v;
__constant__  fortReal8 *dc_G_eta;
__constant__  fortReal8 *dc_G_u;
__constant__  fortReal8 *dc_G_v;
__constant__  fortReal8 *dc_SWM_eta;
__constant__  fortReal8 *dc_SWM_u;
__constant__  fortReal8 *dc_SWM_v;
__constant__  fortReal8 *dc_F_eta;
__constant__  fortReal8 *dc_F_x;
__constant__  fortReal8 *dc_F_y;

//=============================================================================
void cuOptimizeConstParams()
{
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_dt), &hConstParams.dt, sizeof(fortReal8)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_AB_C1), &hConstParams.AB_C1, sizeof(fortReal8)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_AB_C2), &hConstParams.AB_C2, sizeof(fortReal8)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_Nx), &hConstParams.Nx, sizeof(fortInteger)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_Ny), &hConstParams.Ny, sizeof(fortInteger)) );
}

//=============================================================================
void cuOptimizeVarParams()
{
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_NG0), &hVarParams.NG0, sizeof(int)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_NG0m1), &hVarParams.NG0m1, sizeof(int)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_N0), &hVarParams.N0, sizeof(int)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_N0p1), &hVarParams.N0p1, sizeof(int)) );
}

//=============================================================================
void cuOptimizeDevMem()
{
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_ocean_eta), 	&hDevMem.ocean_eta, 	sizeof(fortInteger*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_ocean_u), 	&hDevMem.ocean_u, 	sizeof(fortInteger*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_ocean_v), 	&hDevMem.ocean_v, 	sizeof(fortInteger*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_Coef_eta), 	&hDevMem.SWM_Coef_eta, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_Coef_u), 	&hDevMem.SWM_Coef_u, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_Coef_v), 	&hDevMem.SWM_Coef_v, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_impl_eta), 	&hDevMem.impl_eta, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_impl_u), 	&hDevMem.impl_u, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_impl_v), 	&hDevMem.impl_v, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_G_eta), 		&hDevMem.G_eta, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_G_u), 		&hDevMem.G_u, 		sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_G_v), 		&hDevMem.G_v, 		sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_eta), 	&hDevMem.SWM_eta, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_u), 		&hDevMem.SWM_u, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_SWM_v), 		&hDevMem.SWM_v, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_F_eta), 		&hDevMem.F_eta, 	sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_F_x), 		&hDevMem.F_x, 		sizeof(fortReal8*)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dc_F_y), 		&hDevMem.F_y, 		sizeof(fortReal8*)) );
}

//=============================================================================
// Makros to access arrays in fortran style
// Array access is 'circular', meaning index violations non-existent
// Constant input:
#define ocean_eta(i,j) 		dc_ocean_eta[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define ocean_u(i,j) 		dc_ocean_u[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define ocean_v(i,j) 		dc_ocean_v[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define impl_eta(i,j)		dc_impl_eta[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define impl_u(i,j)		dc_impl_u[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define impl_v(i,j)		dc_impl_v[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define SWM_Coef_eta(i,j,k) 	dc_SWM_Coef_eta[(k) * dc_Nx * 9 + ((j) % dc_Ny) * 9 + (i)]
#define SWM_Coef_u(i,j,k) 	dc_SWM_Coef_u[(k) * dc_Nx * 11 + ((j) % dc_Ny) * 11 + (i)]
#define SWM_Coef_v(i,j,k) 	dc_SWM_Coef_v[(k) * dc_Nx * 11 + ((j) % dc_Ny) * 11 + (i)]

// Externally changed input:
#define F_eta(i,j) 		dc_F_eta[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define F_x(i,j) 		dc_F_x[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define F_y(i,j) 		dc_F_y[((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]

// Cuda Output:
#define G_eta(i,j,k) 		dc_G_eta[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define G_u(i,j,k) 		dc_G_u[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define G_v(i,j,k) 		dc_G_v[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define SWM_eta(i,j,k)		dc_SWM_eta[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define SWM_u(i,j,k) 		dc_SWM_u[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]
#define SWM_v(i,j,k) 		dc_SWM_v[(k) * dc_Nx * dc_Ny + ((j) % dc_Ny) * dc_Nx + ((i) % dc_Nx)]

//=============================================================================
// Kernels for adamBashforth timestep()
//=============================================================================
__global__ void adamBashforth_eta(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_eta(i, j) == 1 )
	{
		G_eta(i, j, dc_NG0) = 
				SWM_eta(i, j, dc_N0) 		* SWM_Coef_eta(0,i,j) +
				SWM_eta(i + 1, 	j, dc_N0)	* SWM_Coef_eta(1,i,j) +
				SWM_eta(i - 1, 	j, dc_N0) 	* SWM_Coef_eta(2,i,j) +
				SWM_eta(i, j + 1, dc_N0) 	* SWM_Coef_eta(3,i,j) +
				SWM_eta(i, j - 1, dc_N0) 	* SWM_Coef_eta(4,i,j) +
				SWM_u(	i + 1, 	j, dc_N0) 	* SWM_Coef_eta(5,i,j) +
				SWM_u(	i, j, dc_N0) 		* SWM_Coef_eta(6,i,j) +
				SWM_v(	i, j + 1, dc_N0) 	* SWM_Coef_eta(7,i,j) +
				SWM_v(	i, j, dc_N0) 		* SWM_Coef_eta(8,i,j) +
				F_eta(	i, j);
		  
		SWM_eta(i, j, dc_N0p1) =
				(SWM_eta(i, j, dc_N0) + dc_dt * (dc_AB_C1 * G_eta(i, j, dc_NG0) - dc_AB_C2 * G_eta(i, j, dc_NG0m1)))/impl_eta(i,j);
	}
}

//=============================================================================
__global__ void adamBashforth_u(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_u(i, j) == 1 )
	{
		G_u(i, j, dc_NG0) = 
				SWM_u(	i, 	j, dc_N0) 	* SWM_Coef_u(0, i, j) +
				SWM_u(	i + 1, 	j, dc_N0) 	* SWM_Coef_u(1, i, j) +
				SWM_u(	i - 1, 	j, dc_N0) 	* SWM_Coef_u(2, i, j) +
				SWM_u(	i, 	j + 1, dc_N0) 	* SWM_Coef_u(3, i, j) +
				SWM_u(	i, 	j - 1, dc_N0) 	* SWM_Coef_u(4, i, j) +
				SWM_v(	i, 	j, dc_N0) 	* SWM_Coef_u(5, i, j) +
				SWM_v(	i - 1,	j, dc_N0) 	* SWM_Coef_u(6, i, j) +
				SWM_v(	i - 1, 	j + 1, dc_N0) 	* SWM_Coef_u(7, i, j) +
				SWM_v(	i, 	j + 1, dc_N0) 	* SWM_Coef_u(8, i, j) +
				SWM_eta(i, 	j, dc_N0)	* SWM_Coef_u(9,i, j) +
				SWM_eta(i - 1, 	j, dc_N0)	* SWM_Coef_u(10,i, j) +
				F_x(i, j);
		
		SWM_u(i, j, dc_N0p1) =
				(SWM_u(i, j, dc_N0) + dc_dt * (dc_AB_C1 * G_u(i, j, dc_NG0) - dc_AB_C2 * G_u(i, j, dc_NG0m1)))/impl_u(i,j);
	}
}

//=============================================================================
__global__ void adamBashforth_v(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_v(i, j) == 1 )
	{
		G_v(i, j, dc_NG0) = 
				SWM_v(	i, 	j, dc_N0) 	* SWM_Coef_v(0, i, j) +
				SWM_v(	i + 1, 	j, dc_N0) 	* SWM_Coef_v(1, i, j) +
				SWM_v(	i - 1, 	j, dc_N0) 	* SWM_Coef_v(2, i, j) +
				SWM_v(	i, 	j + 1, dc_N0) 	* SWM_Coef_v(3, i, j) +
				SWM_v(	i, 	j - 1, dc_N0) 	* SWM_Coef_v(4, i, j) +
				SWM_u(	i + 1, 	j - 1, dc_N0) 	* SWM_Coef_v(5, i, j) +
				SWM_u(	i, 	j - 1, dc_N0) 	* SWM_Coef_v(6, i, j) +
				SWM_u(	i, 	j, dc_N0) 	* SWM_Coef_v(7, i, j) +
				SWM_u(	i + 1, 	j, dc_N0) 	* SWM_Coef_v(8, i, j) +
				SWM_eta(i, 	j, dc_N0)	* SWM_Coef_v(9,i, j) +
				SWM_eta(i, 	j - 1, dc_N0)	* SWM_Coef_v(10,i, j) +
				F_y(i, j);
		
		SWM_v(i, j, dc_N0p1) =
				(SWM_v(i, j, dc_N0) + dc_dt * (dc_AB_C1 * G_v(i, j, dc_NG0) - dc_AB_C2 * G_v(i, j, dc_NG0m1)))/impl_v(i,j);
	}
}

//=============================================================================
// Kernels for euler forward iteration
//=============================================================================
__global__ void eulerForward_eta(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_eta(i, j) == 1 )
	{
		G_eta(i, j, dc_NG0) = 
				SWM_eta(i, j, dc_N0) 	* SWM_Coef_eta(0,i,j) +
				SWM_eta(i + 1, j, dc_N0) * SWM_Coef_eta(1,i,j) +
				SWM_eta(i - 1, j, dc_N0) * SWM_Coef_eta(2,i,j) +
				SWM_eta(i, j + 1, dc_N0)* SWM_Coef_eta(3,i,j) +
				SWM_eta(i, j - 1, dc_N0) * SWM_Coef_eta(4,i,j) +
				SWM_u(i + 1, j, dc_N0) 	* SWM_Coef_eta(5,i,j) +
				SWM_u(i, j, dc_N0)	* SWM_Coef_eta(6,i,j) +
				SWM_v(i, j + 1, dc_N0) 	* SWM_Coef_eta(7,i,j) +
				SWM_v(i, j, dc_N0) 	* SWM_Coef_eta(8,i,j) +
				F_eta(i, j);
		
		SWM_eta(i, j, dc_N0p1) = (SWM_eta(i, j, dc_N0) + dc_dt * G_eta(i, j, dc_NG0))/impl_eta(i,j);
	}
}

//=============================================================================
__global__ void eulerForward_u(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_u(i, j) == 1 )
	{
		G_u(i, j, dc_NG0) = 
				SWM_u(i, j, dc_N0) 	* SWM_Coef_u(0,i,j) +
				SWM_u(i + 1, j, dc_N0) 	* SWM_Coef_u(1,i,j) +
				SWM_u(i - 1, j, dc_N0) 	* SWM_Coef_u(2,i,j) +
				SWM_u(i, j + 1, dc_N0)	* SWM_Coef_u(3,i,j) +
				SWM_u(i, j - 1, dc_N0) 	* SWM_Coef_u(4,i,j) +
				SWM_v(i, j, dc_N0) 	* SWM_Coef_u(5,i,j) +
				SWM_v(i - 1,j, dc_N0) 	* SWM_Coef_u(6,i,j) +
				SWM_v(i - 1, j + 1, dc_N0) * SWM_Coef_u(7,i,j) +
				SWM_v(i, j + 1, dc_N0) 	* SWM_Coef_u(8,i,j) +
				SWM_eta(i, j, dc_N0) 	* SWM_Coef_u(9,i,j) +
				SWM_eta(i - 1, j, dc_N0) * SWM_Coef_u(10,i,j) +
				F_x(i, j);
		
		SWM_u(i, j, dc_N0p1) = (SWM_u(i, j, dc_N0) + dc_dt * G_u(i, j, dc_NG0))/impl_u(i,j);
	}
}    

//=============================================================================
__global__ void eulerForward_v(/*struct constExecutionParameters const * const constParams, struct varExecutionParameters const * const varParams, struct devMemory const * const devMem*/)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
	if( i < dc_Nx && j < dc_Ny && ocean_v(i, j) == 1 )
	{
		G_v(i, j, dc_NG0) = 
				SWM_v(i, j, dc_N0) 	* SWM_Coef_v(0,i,j) +
				SWM_v(i + 1, j, dc_N0) 	* SWM_Coef_v(1,i,j) +
				SWM_v(i - 1, j, dc_N0) 	* SWM_Coef_v(2,i,j) +
				SWM_v(i, j + 1, dc_N0)	* SWM_Coef_v(3,i,j) +
				SWM_v(i, j - 1, dc_N0) 	* SWM_Coef_v(4,i,j) +
				SWM_u(i + 1, j - 1, dc_N0) * SWM_Coef_v(5,i,j) +
				SWM_u(i,j - 1, dc_N0) 	* SWM_Coef_v(6,i,j) +
				SWM_u(i, j, dc_N0) 	* SWM_Coef_v(7,i,j) +
				SWM_u(i + 1, j, dc_N0) 	* SWM_Coef_v(8,i,j) +
				SWM_eta(i, j, dc_N0) 	* SWM_Coef_v(9,i,j) +
				SWM_eta(i, j - 1, dc_N0) 	* SWM_Coef_v(10,i,j) +
				F_y(i, j);
		
		SWM_v(i, j, dc_N0p1) = (SWM_v(i, j, dc_N0) + dc_dt * G_v(i, j, dc_NG0))/impl_v(i,j);
	}
}
//=============================================================================
